
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
#include<math.h>

void sum_arrays_xy(double * xref, double * yref, int Nx, int Ny, long int * axes, float * image)
{
    // i is the x axis (row)
    // j is the y axis (col)
    int i, j;
    double mean=0.;
    for (i=0; i < Nx; i++)
    {
        if (i < axes[1]) 
        {
            for (j=0; j < axes[0]; j++)
            {
                xref[i] +=  (double) image[j*i + j];
                mean += (double) image[j*i + j];
            }
        }
        else xref[i] = 0.; // zero padding
    }
    for (j=0; j < axes[1]; j++)
        xref[j] = xref[j] - mean/axes[0];
    
    printf("\nMean 1 : %f", mean/axes[1]);
    mean = 0.;
    for (j=0; j < Ny ; j++)
    {
        if (j < axes[0]) 
        {
            for (i=0; i < axes[1]; i++)
            {
                yref[j] +=  (double) image[j*i + j];
                mean += (double) image[j*i + j];
            }
        }
        else yref[j] = 0.; // zero padding
    }
    for (i=0; i < axes[0]; i++)
        yref[i] = yref[i] - mean/axes[0];
    printf("Mean 2 : %f", mean/axes[1]);

}