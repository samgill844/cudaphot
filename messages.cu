
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

#if defined (_OPENMP) && !defined(_OPENACC)
#  include <omp.h>
#endif

#ifndef M_PI
    #define M_PI 3.14159265358979323846
#endif

void welcome()
{

    printf("\n-------------------------------------------------");
    printf("\n-              CUDAPHOT V0.1                    -");
    printf("\n-           samgill844@gmail.com                -");
    printf("\n-                                               -");
    int nDevices;
    hipGetDeviceCount(&nDevices);
    printf("\n- Summary                                       -");
    printf("\n-           GPU acceleration [%s]             -", nDevices ? "True" : "False");
    #if defined (_OPENMP) && !defined(_OPENACC)
        printf("\n-        OpenMP acceleration [True]            -");
    #else
        printf("\n-        OpenMP acceleration [False]            -");
    #endif
    printf("\n-------------------------------------------------");
}

void no_arguments()
{
    printf("\nIncorrect number of arguments specified.");
    printf("\nUsage:");
    printf("\n\tcudaphot [ref image] [files]\n\n\n");
}

void failed_to_load_image(const char * filename)
{
    printf("\n\tUnable to load %s\n\n\n", filename);
}

void failed_to_read_image(const char * filename)
{
    printf("\n\tUnable to read %s\n\n\n", filename);
}

void failed_to_allocate_image(const char * filename, int bitpix)
{
    printf("\n\tUnable to allocate %s with BITPIX: %d\n\n", filename, bitpix);
}


void failed_to_load_reference(const char * filename, long int * naxes, int bitpix, int status)
{
    printf("\nFAILED:  %s (%ld x %ld) with errorcode : %d\n\n", filename, naxes[0], naxes[1], status);
}
